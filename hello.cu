
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

// The __global__ specifier indicates a function that runs on device (GPU). 
// Such function can be called through host code, e.g. the main() function in the example, and is also known as "kernels".

int main() {
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize(); // Wait for GPU to finish
    return 0;
}